//  qlogin -l gpu_host,gpus=1
//  /usr/local/cuda/include/cuda_runtime.h
//  nvcc nbodyCUDA.cu -I/usr/local/cuda/include/ -L/usr/local/cuda/lib64


#include <hip/hip_runtime.h>
#include <stdio.h>

#define POINTS 7500
#define COL 3
#define CLUSTERS 50
#define X 0
#define Y 1
#define FLT_MAX 2.402823e+37

// [x location][y location][associated centroid (0-50)]
int inputPoints[POINTS][COL];
//  [x location][y location]
float xCentroids[CLUSTERS];
float yCentroids[CLUSTERS];

__global__ void parallel(float * xTotal, float * yTotal, float * nTotal, float * xCentroids, float * yCentroids){
  int idx = threadIdx.x;
  float x = xTotal[idx] / nTotal[idx];
  xCentroids[idx] = x;
  float y = yTotal[idx] / nTotal[idx];
  yCentroids[idx] = y;
}

__host__ void cuda_error_check() {
  if(hipPeekAtLastError() != hipSuccess) {
    printf("\nCUDA error: %s\n\n", hipGetErrorString(hipGetLastError()));
    hipDeviceReset();
    //wait_exit();
    exit(1);
  } else {
    printf("\nNo cuda Errors Detected\n\n");
  }
}

void initialAssignCentroids(float xCentroids[], float yCentroids[]) {
  int i = 0;
  for(int x = 1; x <= 5; x++) {
    for(int y = 1; y <= 10; y++) {
      xCentroids[i] = x*11000+10000;
      yCentroids[i] = y*2200+10000;
      i++;
    }
  }
}

void printCentroids(float xCentroids[], float yCentroids[])
{
  int p;
  for(p=0; p<CLUSTERS; p++)
    {
      printf("[centroid]= %d ( %.2f , %.2f )\n", p, xCentroids[p], yCentroids[p]);
    }
}

void printInput(int score[][COL])
{
  int p;
  for(p=0; p<POINTS; p++)
  {
    printf("[%d %d]= %d [%d %d]= %d [centroid]= %d\n", p, X, score[p][X],
      p, Y, score[p][Y], score[p][2]);
  }
}

void readInput(int score[POINTS][COL])
{
  FILE *f=fopen("a3.txt", "r");

  int p;
  for(p=0; p<POINTS; p++)
    {
      fscanf(f, "%d %d", &score[p][X], &score[p][Y]);
      score[p][2] = 0;
    }
}

float pointDistance(int x1, int y1, float x2, float y2) {
  return sqrt(pow(x1-x2, 2)+pow(y1-y2, 2));
}

int assignCentroids(int points[][COL], float xCentroids[], float yCentroids[]) {
  int pointsUpdated = 0;
  for(int p = 0; p < POINTS; p++) {
    float minDist = FLT_MAX;
    int closestCentroidId = 0;
    for(int c = 0; c < CLUSTERS; c++) {
      float distance = pointDistance(points[p][X], points[p][Y], xCentroids[c], yCentroids[c]);
      if(distance < minDist) {
        minDist = distance;
        closestCentroidId = c;

      }

    }
    if(closestCentroidId != points[p][2]) {
      pointsUpdated++;
      points[p][2] = closestCentroidId;
    }
  }
  return pointsUpdated;
}

void moveCentroids(int points[][COL], float xCentroids[], float yCentroids[]) {
  static float xTotal[CLUSTERS];
  static float yTotal[CLUSTERS];
  static float nTotal[CLUSTERS];
  for(int i = 0; i<CLUSTERS; i++){
    xTotal[i] = yTotal[i] = nTotal[i] = 0;
  }

  for(int p = 0; p<POINTS; p++) {
    xTotal[points[p][2]] += points[p][0];
    yTotal[points[p][2]] += points[p][1];
    nTotal[points[p][2]] += 1;
  }

  for(int c = 0; c<CLUSTERS; c++){
    xCentroids[c] = xTotal[c]/nTotal[c];
    yCentroids[c] = yTotal[c]/nTotal[c];
  }
}

void moveCentroidsCUDA(int h_points[][3], float h_xCentroids[], float h_yCentroids[]) {
  static float h_xTotal[CLUSTERS];
  static float h_yTotal[CLUSTERS];
  static float h_nTotal[CLUSTERS];
  for(int i = 0; i<CLUSTERS; i++){
    h_xTotal[i] = h_yTotal[i] = h_nTotal[i] = 0;
  }

  for(int p = 0; p<POINTS; p++) {
    h_xTotal[h_points[p][2]] += h_points[p][0];
    h_yTotal[h_points[p][2]] += h_points[p][1];
    h_nTotal[h_points[p][2]] += 1;
  }

  const int TOTALS_ARRAY_SIZE = CLUSTERS;
  const int TOTALS_ARRAY_BYTES = TOTALS_ARRAY_SIZE * sizeof(float);

  const int CENTROID_ARRAY_SIZE = CLUSTERS;
  const int CENTROID_ARRAY_BYTES = CENTROID_ARRAY_SIZE * sizeof(float);

  float * d_xTotal;
  float * d_yTotal;
  float * d_nTotal;
  float * d_xCentroids;
  float * d_yCentroids;

  hipMalloc((void**) &d_xTotal, TOTALS_ARRAY_BYTES);
  hipMalloc((void**) &d_yTotal, TOTALS_ARRAY_BYTES);
  hipMalloc((void**) &d_nTotal, TOTALS_ARRAY_BYTES);
  hipMalloc((void**) &d_xCentroids, CENTROID_ARRAY_BYTES);
  hipMalloc((void**) &d_yCentroids, CENTROID_ARRAY_BYTES);

  hipMemcpy(d_xTotal, h_xTotal, TOTALS_ARRAY_BYTES, hipMemcpyHostToDevice);
  hipMemcpy(d_yTotal, h_yTotal, TOTALS_ARRAY_BYTES, hipMemcpyHostToDevice);
  hipMemcpy(d_nTotal, h_nTotal, TOTALS_ARRAY_BYTES, hipMemcpyHostToDevice);
  hipMemcpy(d_xCentroids, h_xCentroids, CENTROID_ARRAY_BYTES, hipMemcpyHostToDevice);
  hipMemcpy(d_yCentroids, h_yCentroids, CENTROID_ARRAY_BYTES, hipMemcpyHostToDevice);

  parallel<<<1, CLUSTERS>>>(d_xTotal, d_yTotal, d_nTotal, d_xCentroids, d_yCentroids);
  cuda_error_check();

  hipMemcpy(h_xTotal, d_xTotal, TOTALS_ARRAY_BYTES, hipMemcpyDeviceToHost);
  hipMemcpy(h_yTotal, d_yTotal, TOTALS_ARRAY_BYTES, hipMemcpyDeviceToHost);
  hipMemcpy(h_nTotal, d_nTotal, TOTALS_ARRAY_BYTES, hipMemcpyDeviceToHost);
  hipMemcpy(h_xCentroids, d_xCentroids, CENTROID_ARRAY_BYTES, hipMemcpyDeviceToHost);
  hipMemcpy(h_yCentroids, d_yCentroids, CENTROID_ARRAY_BYTES, hipMemcpyDeviceToHost);

  hipFree(d_xTotal);
  hipFree(d_yTotal);
  hipFree(d_nTotal);
  hipFree(d_xCentroids);
  hipFree(d_yCentroids);
}

int main(int argc, char ** argv) {
  readInput(inputPoints);
  printInput(inputPoints);
  initialAssignCentroids(xCentroids, yCentroids);
  printCentroids(xCentroids, yCentroids);
  int u = 1;
  while(u > 0) {
    u = assignCentroids(inputPoints, xCentroids, yCentroids);
    printf("%d points updated\n", u);
    moveCentroids(inputPoints, xCentroids, yCentroids);
    //moveCentroidsCUDA(inputPoints, xCentroids, yCentroids);
  }
  printCentroids(xCentroids, yCentroids);

	return 0;
}
